#include "hip/hip_runtime.h"
#include "intersections.h"

__host__ __device__ 
float boxIntersectionTest(
    Geom box,
    Ray r,
    glm::vec3 &intersectionPoint,
    glm::vec3 &normal,
    bool &outside)
{
    Ray q;
    q.origin    =                multiplyMV(box.inverseTransform, glm::vec4(r.origin   , 1.0f));
    q.direction = glm::normalize(multiplyMV(box.inverseTransform, glm::vec4(r.direction, 0.0f)));

    float tmin = -1e38f;
    float tmax = 1e38f;
    glm::vec3 tmin_n;
    glm::vec3 tmax_n;
    for (int xyz = 0; xyz < 3; ++xyz)
    {
        float qdxyz = q.direction[xyz];
        /*if (glm::abs(qdxyz) > 0.00001f)*/
        {
            float t1 = (-0.5f - q.origin[xyz]) / qdxyz;
            float t2 = (+0.5f - q.origin[xyz]) / qdxyz;
            float ta = glm::min(t1, t2);
            float tb = glm::max(t1, t2);
            glm::vec3 n;
            n[xyz] = t2 < t1 ? +1 : -1;
            if (ta > 0 && ta > tmin)
            {
                tmin = ta;
                tmin_n = n;
            }
            if (tb < tmax)
            {
                tmax = tb;
                tmax_n = n;
            }
        }
    }

    if (tmax >= tmin && tmax > 0)
    {
        outside = true;
        if (tmin <= 0)
        {
            tmin = tmax;
            tmin_n = tmax_n;
            outside = false;
        }
        intersectionPoint = multiplyMV(box.transform, glm::vec4(getPointOnRay(q, tmin), 1.0f));
        normal = glm::normalize(multiplyMV(box.invTranspose, glm::vec4(tmin_n, 0.0f)));
        return glm::length(r.origin - intersectionPoint);
    }

    return -1;
}

__host__ __device__ 
float sphereIntersectionTest(
    Geom sphere,
    Ray r,
    glm::vec3 &intersectionPoint,
    glm::vec3 &normal,
    bool &outside)
{
    float radius = .5;

    glm::vec3 ro = multiplyMV(sphere.inverseTransform, glm::vec4(r.origin, 1.0f));
    glm::vec3 rd = glm::normalize(multiplyMV(sphere.inverseTransform, glm::vec4(r.direction, 0.0f)));

    Ray rt;
    rt.origin = ro;
    rt.direction = rd;

    float vDotDirection = glm::dot(rt.origin, rt.direction);
    float radicand = vDotDirection * vDotDirection - (glm::dot(rt.origin, rt.origin) - powf(radius, 2));
    if (radicand < 0)
    {
        return -1;
    }

    float squareRoot = sqrt(radicand);
    float firstTerm = -vDotDirection;
    float t1 = firstTerm + squareRoot;
    float t2 = firstTerm - squareRoot;

    float t = 0;
    if (t1 < 0 && t2 < 0)
    {
        return -1;
    }
    else if (t1 > 0 && t2 > 0)
    {
        t = min(t1, t2);
        outside = true;
    }
    else
    {
        t = max(t1, t2);
        outside = false;
    }

    glm::vec3 objspaceIntersection = getPointOnRay(rt, t);

    intersectionPoint = multiplyMV(sphere.transform, glm::vec4(objspaceIntersection, 1.f));
    normal = glm::normalize(multiplyMV(sphere.invTranspose, glm::vec4(objspaceIntersection, 0.f)));
    if (!outside)
    {
        normal = -normal;
    }

    return glm::length(r.origin - intersectionPoint);
}

// ������״����ײ���:
// ����: ����
__host__ __device__ 
float meshIntersectionTest(
    glm::vec3& intersection_point, Geom mesh, Ray r, 
    glm::vec2& texture_coord, glm::vec3& normal, 
    Triangle* triangles, bool& from_outside)
{
    // ����mesh�ϵĹ���
    glm::vec3 ray_origin = multiplyMV(mesh.inverseTransform, glm::vec4(r.origin, 1.0f));
    glm::vec3 ray_direction = glm::normalize(multiplyMV(mesh.inverseTransform, glm::vec4(r.direction, 0.0f)));
    Ray ray{ ray_origin,ray_direction };
    int min_idx = -1;
    float tmin = FLT_MAX;
    glm::vec3 barypos(0.0f), minbarypos(0.0f);
    for (int i = mesh.tri_start_idx; i < mesh.tri_start_idx + mesh.n_tris; ++i) {
        auto& t = triangles[i];
        // �������� barypos:
        // The baryPosition output uses barycentric coordinates for the x and y components.The z component is the scalar factor for ray.
        // That is, 1.0 - baryPosition.x - baryPosition.y = actual z barycentric coordinate
        if (glm::intersectRayTriangle( 
            ray.origin, ray.direction, t.vertices[0], t.vertices[1], t.vertices[2], barypos
        )) {
            if (barypos.z > 0.0f && barypos.z < tmin) {
                min_idx = i;
                tmin = barypos.z;
                minbarypos = barypos;
            }
        }
    }
    if (min_idx == -1)
        return -1.0f;
    // �������棬�����������꣬��Triangle��vertices_texture_coord�ֶ�
    auto actual_z = 1.0f - minbarypos.x - minbarypos.y;
    auto& tri = triangles[min_idx];
    normal = actual_z * tri.vertex_normals[0] + tri.vertex_normals[1] + tri.vertex_normals[2];
    normal = glm::normalize(normal);
    texture_coord = actual_z * tri.vertices_texture_coord[0] + tri.vertices_texture_coord[1] + tri.vertices_texture_coord[2];
    return tmin;
}
// ���񣺰�Χ��
__host__ __device__
float IntersectBVH(glm::vec3& intersection_point, Geom mesh, Ray r, glm::vec3& normal, Triangle* triangles, bool& from_outside)
{
    // TODO: 
    return -1.0f;
}